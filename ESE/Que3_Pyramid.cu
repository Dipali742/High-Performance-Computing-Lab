#include "hip/hip_runtime.h"
%%writefile Que3_Pyramid.cu

#include<stdio.h>
#include<iostream>
using namespace std;

__global__ void Pyramid(int *gpu_num,long int *gpu_res)
{
  int space;
  for(int i = 1, k = 0; i <= *gpu_num; ++i, k = 0)
    {
        for(space = 1; space <= *gpu_num-i; ++space)
        {
            printf("  ");
        }

        while(k != 2*i-1)
        {
            
            printf("* ");
            ++k;
        }
        printf("\n");
    } 
}

int main()
{
  int Number=6;  
  int *dev_number;
  long int *res, result; 
  
  printf("\nHeight of Pyramid : %d\n",Number);
  
  
  hipMalloc((void**)&dev_number,sizeof(int));
  hipMalloc((void**)&res,sizeof(long int));
  
  hipMemcpy(dev_number,&Number,sizeof(int),hipMemcpyHostToDevice);
 
  Pyramid<<<1,1>>>(dev_number,res);

  hipMemcpy(&result,res,sizeof(long int),hipMemcpyDeviceToHost);

 
  return 0; 
}